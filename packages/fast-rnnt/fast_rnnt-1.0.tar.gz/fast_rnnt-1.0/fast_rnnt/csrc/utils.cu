#include "hip/hip_runtime.h"
/**
 * Copyright      2022  Xiaomi Corporation (authors: Wei Kang)
 *
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "fast_rnnt/csrc/utils.h"

namespace fast_rnnt  {

void MonotonicLowerBound(torch::Tensor &src) {
  TORCH_CHECK(src.dim() == 1, "Only support one dimension tensor");
  TORCH_CHECK(src.scalar_type() == torch::kLong, "Only support LongTensor");
  TORCH_CHECK(src.is_contiguous(), "Expected to be contiguous");
  int32_t dim = src.numel();
  if (src.device().type() == torch::kCPU) {
    int64_t min_value = std::numeric_limits<int64_t>::max();
    int64_t *src_data = src.data_ptr<int64_t>();
    for (int32_t i = dim - 1; i >= 0; --i) {
      min_value = std::min(src_data[i], min_value);
      src[i] = min_value;
    }
  } else {
#ifdef FT_WITH_CUDA
    TORCH_CHECK(src.device().is_cuda());
    internal::MinOp<int64_t> min_op;
    auto src_data = src.data_ptr<int64_t>();
    internal::ConstReversedPtr<int64_t> src_ptr =
        internal::ConstReversedPtr<int64_t>(src_data, dim);
    internal::ReversedPtr<int64_t> dest_ptr =
        internal::ReversedPtr<int64_t>(src_data, dim);
    // The first time is to determine temporary device storage requirements.
    std::size_t temp_storage_bytes = 0;
    auto s = hipcub::DeviceScan::InclusiveScan(nullptr, temp_storage_bytes,
                                            src_ptr, dest_ptr, min_op, dim);
    TORCH_CHECK(s == hipSuccess, hipGetErrorString(s));

    auto d_temp = torch::empty({static_cast<int64_t>(temp_storage_bytes)},
                               torch::dtype(torch::kInt8).device(src.device()));
    int8_t *d_temp_storage = d_temp.data_ptr<int8_t>();
    s = hipcub::DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes,
                                       src_ptr, dest_ptr, min_op, dim);
    TORCH_CHECK(s == hipSuccess, hipGetErrorString(s));
#else
    TORCH_CHECK(false, "Please build with -DFT_WITH_CUDA=ON");
#endif  // FT_WITH_CUDA
  }
}

} // namespace fast_rnnt

